#include <cstdlib>
#include <iomanip>
#include <iostream>

#include <mpi.h>

#define CHECK_ERROR(call)                                                                                         \
do {                                                                                                              \
    hipError_t error = call;                                                                                     \
    if (error != hipSuccess) {                                                                                   \
        std::cout << "CUDA Error: " << __FILE__ << ", " << __LINE__ << ", " << hipGetErrorString(error) << "\n"; \
        exit(EXIT_FAILURE);                                                                                       \
    }                                                                                                             \
} while(0)

int main(int argc, char** argv) {

    // Initialize MPI
    MPI_Init(&argc, &argv);

    int rank = -1;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    
    int size = -1;
    MPI_Comm_size(MPI_COMM_WORLD, &size);


    // Check that program uses only two MPI ranks
    if (size != 2) {
        if (rank == 0) {
            std::cout << "This program requires exactly 2 MPI ranks, you used " << size << "\n";
            std::cout << "Aborting.\n";
        }
        MPI_Finalize();
        exit(EXIT_FAILURE);
    }

    MPI_Status status;

    // Map devices to MPI ranks
    int device_count = 0;
    CHECK_ERROR( hipGetDeviceCount(&device_count) );
    CHECK_ERROR( hipSetDevice(rank % device_count) );

    // Loop on message sizes ranging from 8B to 1GB
    for (size_t shift = 0; shift < 27; ++shift) {

        // Allocate and initialize buffer
        const int N = 1 << shift; // MPI wants an int here
        double* buffer = new double[N];

        for (size_t i = 0; i < N; ++i) {
            buffer[i] = rand() / static_cast<double>(RAND_MAX);
        }

        // Copy buffer to GPU
        double* gpu_buffer;
        CHECK_ERROR( hipMalloc(&gpu_buffer, N * sizeof(double)) );
        CHECK_ERROR( hipMemcpy(gpu_buffer, buffer, N * sizeof(double), hipMemcpyHostToDevice) );

        // Tags for message exchange
        constexpr int tag1 = 10;
        constexpr int tag2 = 20;

        constexpr size_t warm_up_reps = 5;
        constexpr size_t timed_reps = 50;

        // Warm-up loop
        for (size_t i = 0; i < warm_up_reps; ++i) {
            if (rank == 0) {
                MPI_Send(gpu_buffer, N, MPI_DOUBLE, 1, tag1, MPI_COMM_WORLD);
                MPI_Recv(gpu_buffer, N, MPI_DOUBLE, 1, tag2, MPI_COMM_WORLD, &status);
            } else if (rank == 1) {
                MPI_Recv(gpu_buffer, N, MPI_DOUBLE, 0, tag1, MPI_COMM_WORLD, &status);
                MPI_Send(gpu_buffer, N, MPI_DOUBLE, 0, tag2, MPI_COMM_WORLD);
            }
        }

        // Measure ping pong bandwidth
        double start_time, stop_time, elapsed_time;
        start_time = MPI_Wtime();

        for (size_t i = 0; i < timed_reps; ++i) {
            if (rank == 0) {
                MPI_Send(buffer, N, MPI_DOUBLE, 1, tag1, MPI_COMM_WORLD);
                MPI_Recv(buffer, N, MPI_DOUBLE, 1, tag2, MPI_COMM_WORLD, &status);
            } else if (rank == 1) {
                MPI_Recv(buffer, N, MPI_DOUBLE, 0, tag1, MPI_COMM_WORLD, &status);
                MPI_Send(buffer, N, MPI_DOUBLE, 0, tag2, MPI_COMM_WORLD);
            }

        }

        stop_time = MPI_Wtime();
        elapsed_time = stop_time - start_time;

        // Print measured bandwidth
        size_t message_size_bytes = N * sizeof(double);
        size_t bytes_in_gigabyte = 1 << 30;

        double message_size_gigabytes = message_size_bytes / static_cast<double>(bytes_in_gigabyte);
        double avg_time_per_transfer = elapsed_time / static_cast<double>(2 * timed_reps);
        double bandwidth = message_size_gigabytes / avg_time_per_transfer;

        if (rank == 0) {
            std::cout << "Transfer size (B): " << std::setw(10) << message_size_bytes
                      << ", Transfer time (s): " << std::setw(15) << std::setprecision(9) << avg_time_per_transfer
                      << ", Bandwidth (GB/s): " << std::setw(15) << std::setprecision(9) << bandwidth << "\n";
        }

        // Finalize loop iteration
        hipFree(gpu_buffer);
        free(buffer);

    } // end message size loop
    
    MPI_Finalize();
    return EXIT_SUCCESS;
}
